#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>


static void CatchError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}
#define CATCH_ERROR( err ) (CatchError( err, __FILE__, __LINE__ ))


int* find_multipliers(long long int N)
{
    int k;
    int* temp = (int*)malloc(100);
    for (int i = 2; i * i <= N; i++)
    {
        // if composite number:
        if (N % i == 0)
        {
            temp[0] = i;
            printf("%d^", i);
            // calc how many same numbers in composite num
            for (k = 0; N % i == 0; k++)
            {
                temp[k + 1] = k;
                N /= i;
            }
            printf("%d ", k);
        }
    }
    if (N > 1) {
        temp[0] = N;
        printf("%d ", N);
    };
    printf("\n");
    return temp;
}


// func for CPU calculations
void my_find(int** res, long long int n) {
    long long int tid = 0;
    
    while (tid<n)
    {
        res[tid] = find_multipliers(tid);
        tid++;    }
}


__device__ int* find_multipliers_gpu(long long int N)
{
    int k;
    int* temp = (int*)malloc(100);
    for (int i = 2; i * i <= N; i++)
    {
        // if composite number:
        if (N % i == 0)
        {
            printf("%d^", i);
            temp[0] = i;
            // calc how many same numbers in composite num
            for (k = 0; N % i == 0; k++)
            {
                N /= i;
            }
            temp[k + 1] = k;
            printf("%d ", k);
        }
    }
    if (N > 1) 
    {
        temp[0] = N;
        printf("%d ", N);
    }
    printf("\n");
    return temp;
}


__global__ void kernel(int **res, long long int n)
{
    long long int tid = blockIdx.x * blockDim.x + threadIdx.x;
    long long int offset = blockDim.x * gridDim.x;

    
    while (tid < n) {
        res[tid] = find_multipliers_gpu(tid);
        tid += offset;
    }
}



int main()
{
    long long int n = 1024;


    int count;
    CATCH_ERROR(hipGetDeviceCount(&count));
    if (count == 0) {
        printf("there is no cuda device");
        return -1;
    }


    // for timer
    hipEvent_t time_of_start, time_of_end;
    float res_timer_gpu;
    CATCH_ERROR(hipEventCreate(&time_of_start));
    CATCH_ERROR(hipEventCreate(&time_of_end));


    int** res = (int**)malloc(n * 1000 * sizeof(int*));

    int** res_dev;
    CATCH_ERROR(hipMalloc(&res_dev, n * 1000 * sizeof(int*)));
    CATCH_ERROR(hipEventRecord(time_of_start)); // start of timer (GPU)

    kernel << <256, 256 >> > (res_dev, n);

    CATCH_ERROR(hipEventRecord(time_of_end)); // end of timer (GPU)
    CATCH_ERROR(hipEventSynchronize(time_of_end));
    
    CATCH_ERROR(hipEventElapsedTime(&res_timer_gpu, time_of_start, time_of_end));

    CATCH_ERROR(hipEventDestroy(time_of_start));
    CATCH_ERROR(hipEventDestroy(time_of_end));

    CATCH_ERROR(hipMemcpy(res, res_dev, n * sizeof(int), hipMemcpyDeviceToHost));

    CATCH_ERROR(hipFree(res_dev));


    free(res);

    res = (int**)malloc(n * n * sizeof(int));
    double res_timer_cpu = 0.0;

    clock_t begin = clock();

    my_find(res, n);

    clock_t end = clock();
        
    res_timer_cpu += (double)(end - begin) / CLOCKS_PER_SEC;

    printf("Results by CPU: %f\n", res_timer_cpu * 1000);

    printf("Results by GPU: %f\n", res_timer_gpu);

    free(res);
    return 0;
}
