﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>



static void CatchError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}
#define CATCH_ERROR( err ) (CatchError( err, __FILE__, __LINE__ ))


__global__ void kernel(double *res, double *array_degrees, long long int n)
{
    long long int tid = blockIdx.x * blockDim.x + threadIdx.x;
    long long int offset = blockDim.x * gridDim.x;
    
    while (tid < n) {
        // writing to array result of calculations with conversion to degrees
        res[tid] = cosf( array_degrees[tid%90] * 3.141592653589 / 180 );
        tid += offset;
    }
}

void my_cos(double *res, double *array_degrees, long long int n) {
    long long int tid = 0;

    while (tid < n) {
        // writing to array result of calculations with conversion to degrees
        res[tid] = cosf(array_degrees[tid % 90] * 3.141592653589 / 180);
        tid += 1;
    }
}

int main()
{
    int count;
    CATCH_ERROR( hipGetDeviceCount( &count ) );
    if (count == 0) {
        printf("there is no cuda device");
        return -1;
    }

    long long int n = 1000000;

    // for timer
    hipEvent_t time_of_start, time_of_end;
    float res_timer_gpu;
    CATCH_ERROR( hipEventCreate( &time_of_start ) );
    CATCH_ERROR( hipEventCreate( &time_of_end ) );
    


    double array_degrees[90];    // array with degrees
    for (int i = 0; i < 90; i++) {  //  from 0 to 90
        array_degrees[i] = i;
    }

    double *res = (double*)malloc(n * sizeof(double));
    double *ar_d_dev, *res_dev;
    
    CATCH_ERROR( hipMalloc( &res_dev, n * sizeof(double) ) );
    CATCH_ERROR( hipMalloc( &ar_d_dev, 90 * sizeof(double) ) );

    CATCH_ERROR( hipMemcpy( ar_d_dev, array_degrees, 90 * sizeof(double), hipMemcpyHostToDevice ) );
    

    CATCH_ERROR( hipEventRecord( time_of_start ) ); // start of timer (GPU)

    kernel <<<256,256>>>(res_dev, ar_d_dev, n);

    CATCH_ERROR( hipEventRecord( time_of_end )); // end of timer (GPU)
    CATCH_ERROR( hipEventSynchronize( time_of_end ) );
    
    CATCH_ERROR( hipEventElapsedTime( &res_timer_gpu, time_of_start, time_of_end ) );
    
    CATCH_ERROR( hipEventDestroy( time_of_start ) );
    CATCH_ERROR( hipEventDestroy( time_of_end ) );

    CATCH_ERROR( hipMemcpy( res, res_dev, n * sizeof(double), hipMemcpyDeviceToHost ) );
    
    CATCH_ERROR( hipFree( ar_d_dev ) );
    CATCH_ERROR( hipFree( res_dev ) );

    for (long long i = 0; i < n; i++) {
        printf("%f\n", res[i]);
    }

    free(res);

    res = (double*)malloc(n * sizeof(double));
    double res_timer_cpu = 0.0;

    clock_t begin = clock();
    
    my_cos(res, array_degrees, n);

    clock_t end = clock();

    res_timer_cpu += (double)(end - begin) / CLOCKS_PER_SEC;

    printf("Results by CPU: %f\n", res_timer_cpu*1000);

    printf("Results by GPU: %f\n", res_timer_gpu);
    
    free(res);
    return 0;
}